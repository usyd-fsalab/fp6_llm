#include "hip/hip_runtime.h"
#include "include/kernel_matmul.cuh"
#include "include/kernel_reduction.cuh"
#include "utils/weight_prepacking.h"
#include "utils/weight_dequant.h"
#include "utils/weight_quant.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

template<typename TilingConfig, typename OutputDataType, int EXPONENT, int MANTISSA>
static void Kernel_Ex(hipStream_t    stream,
                      const uint4     *Weight,
                      const half      *Scales,
                      const half      *B,
                      OutputDataType  *C,
                      const size_t    M_Global,
                      const size_t    N_Global,
                      const size_t    K_Global, 
                      int             Split_K) 
{
    #ifdef DEBUG_MODE
        printf("\n");
        printf("Launcher.cu->Kernel_Ex():\n");
        printf("M: %d, N: %d, K: %d, SplitK: %d\n", M_Global, N_Global, K_Global, Split_K);
        printf("TILE_M: %d, TILE_K: %d, TILE_N: %d\n", TilingConfig::TILE_M, TilingConfig::TILE_K, TilingConfig::TILE_N);
    #endif
    static size_t SHMEM_SZ = max(TilingConfig::SMEM_SIZE_B_TILE+SMEM_SIZE_PER_TB_A_TILE, TilingConfig::SMEM_SIZE_C_TILE);
    hipFuncSetAttribute(reinterpret_cast<const void*>(QUANT_GEMM_Kernel<TilingConfig), OutputDataType, EXPONENT, MANTISSA>, hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ);
    size_t  dimN = (N_Global-1) / TilingConfig::TILE_N + 1;
    size_t  dimM = M_Global * Split_K / TilingConfig::TILE_M;
    dim3    GridDim(dimN, dimM, 1);
    dim3    BlockDim(WARP_SIZE * TilingConfig::BLOCK_WARPS, 1, 1);
    //
    #ifdef DEBUG_MODE
        printf("GridDim.x: %d, GridDim.y: %d, GridDim.z: %d, BlockDim.x: %d, BlockDim.y: %d, BlockDim.z: %d SHMEM_SZ: %d\n",
                GridDim.x, GridDim.y, GridDim.z, BlockDim.x, BlockDim.y, BlockDim.z, SHMEM_SZ);
        printf("\n");
    #endif
    QUANT_GEMM_Kernel<TilingConfig, OutputDataType, EXPONENT, MANTISSA><<<GridDim, BlockDim, SHMEM_SZ, stream>>>
                    (Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);
}

template<int EXPONENT, int MANTISSA>
hipError_t fpx_linear_kernel(hipStream_t    stream,
                              const uint4     *Weight,
                              const half      *Scales,
                              const half      *B,
                              half            *C,
                              const size_t    M_Global,
                              const size_t    N_Global,
                              const size_t    K_Global, 
                              float           *Reduction_Workspace,  // Reduction_Workspace_Size = Split_K * M_Global * N_Global * sizeof(fp32)
                              int             Split_K)
{
    assert(M_Global % 256 == 0);
    assert(K_Global % 64 == 0);
    assert(N_Global>0);

    // Work around to support more N shapes:
    size_t N_PowerOf2;
    if(N_Global>0 &&  N_Global<=8)      N_PowerOf2 = 8;
    if(N_Global>8 &&  N_Global<=16)     N_PowerOf2 = 16;
    if(N_Global>16 && N_Global<=32)     N_PowerOf2 = 32;
    if(N_Global>32 && N_Global<=64)     N_PowerOf2 = 64;
    if(N_Global>64 && N_Global<=128)    N_PowerOf2 = 128;
    if(N_Global>128)                    N_PowerOf2 = ((N_Global-1)/128+1) * 128;

    if (Split_K == 1) {
        switch (N_PowerOf2) {
            case 8:     Kernel_Ex<TilingConfig<4, 1, 1>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 16:    Kernel_Ex<TilingConfig<4, 1, 2>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 32:    Kernel_Ex<TilingConfig<4, 1, 4>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 64:    Kernel_Ex<TilingConfig<4, 1, 8>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            case 128:   Kernel_Ex<TilingConfig<4, 1, 8>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
            default:    if (N_PowerOf2 % 128 != 0) {
                            printf("FP6LLM_API Error: Unsupported N dimension %d!\n", N_PowerOf2);
                            return hipErrorUnknown;
                        }
                        Kernel_Ex<TilingConfig<4, 1, 8>, half, EXPONENT, MANTISSA>(stream, Weight, Scales, B, C, M_Global, N_Global, K_Global, Split_K);  break;
        }
    }
    else {
        switch (N_PowerOf2) {
            case 8:     Kernel_Ex<TilingConfig<4, 1, 1>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 16:    Kernel_Ex<TilingConfig<4, 1, 2>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 32:    Kernel_Ex<TilingConfig<4, 1, 4>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 64:    Kernel_Ex<TilingConfig<4, 1, 8>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            case 128:   Kernel_Ex<TilingConfig<4, 1, 8>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
            default:    if (N_PowerOf2 % 128 != 0) {
                            printf("FP6LLM_API Error: Unsupported N dimension %d!\n", N_PowerOf2);
                            return hipErrorUnknown;
                        }
                        Kernel_Ex<TilingConfig<4, 1, 8>, float, EXPONENT, MANTISSA>(stream, Weight, Scales, B, Reduction_Workspace, M_Global, N_Global, K_Global, Split_K);  break;
        }
        // Reduction for SplitK
        dim3 GridDim((M_Global * N_Global) / REDUCTION_ELEMENT_PER_THREADBLOCK, 1, 1);
        dim3 BlockDim(WARP_SIZE, 1, 1);
        SplitK_Reduction<<<GridDim, BlockDim, 0, stream>>>(C, Reduction_Workspace, M_Global, N_Global, Split_K);
    }
    return hipGetLastError();
}

hipError_t fp6_linear_kernel(
    hipStream_t    stream,
    const uint4     *Weight,
    const half      *Scales,
    const half      *B,
    half            *C,
    const size_t    M_Global,
    const size_t    N_Global,
    const size_t    K_Global, 
    float           *Reduction_Workspace,
    int             Split_K) {
    //
    return fpx_linear_kernel<3,2>( stream, Weight, Scales, B, C, M_Global, N_Global, K_Global,  Reduction_Workspace, Split_K);
}

hipError_t fp_eXmY_linear_kernel(
    const int       EXPONENT,
    const int       MANTISSA,
    hipStream_t    stream,
    const uint4     *Weight,
    const half      *Scales,
    const half      *B,
    half            *C,
    const size_t    M_Global,
    const size_t    N_Global,
    const size_t    K_Global, 
    float           *Reduction_Workspace,
    int             Split_K) {
    //
    if(EXPONENT==2 && MANTISSA==2)
        return fpx_linear_kernel<2,2>( stream, Weight, Scales, B, C, M_Global, N_Global, K_Global,  Reduction_Workspace, Split_K);
    if(EXPONENT==3 && MANTISSA==2)
        return fpx_linear_kernel<3,2>( stream, Weight, Scales, B, C, M_Global, N_Global, K_Global,  Reduction_Workspace, Split_K);
    printf("QuantLLM_API Error: Unsupported EXPONENT=%d, MANTISSA=%d!\n", EXPONENT, MANTISSA);
    exit(-1);
}

#ifndef NO_PYTORCH
#include <torch/extension.h>
#include <ATen/ATen.h>

/////////////////////////////////////////////////// Old Interface only Supporting FP6 /////////////////////////////////////////////////////////////////////
/*
Computes FP6-FP16 GEMM (PyTorch interface).

[Mathmatical Formula]
Standard definition of linear layer:    Out = In * trans(W), where In, Out, and W are stored in row-major.
After Equivalent transformation    :    trans(Out) = W * trans(In). Note that we do not perform "transpose" during runtime, we instead interpret the In/Out as column-major matrices when calling our CUDA kernel.

[Inputs]
  _in_feats:  tensor of shape [B, IC];                  // half 
  _weights:   int tensor of shape [OC, IC // 16 * 3];   // 3 INT32 words contains 16 FP6 weights.
  _scales:    tensor of shape [OC];                     // half
  splitK:     spliting the MatMul problem along K dimension for higher GPU utilization, default 1.
[Outputs]
  _out_feats: tensor of shape [B, OC];                  // half
*/
torch::Tensor fp6_linear_forward_cuda(
    torch::Tensor _in_feats,
    torch::Tensor _weights,
    torch::Tensor _scales,
    int           splitK=1)
{
    int num_in_feats      = _in_feats.size(0);
    int num_in_channels   = _in_feats.size(1);
    int num_out_channels  = _weights.size(0);
    assert( num_in_channels%64 == 0 );
    assert( (num_in_channels/16*3) == _weights.size(1) );    // Making sure the K dimension is matched.
    //
    int M = num_out_channels;
    int K = num_in_channels;
    int N = num_in_feats;
    // Input Tensors
    auto weight = reinterpret_cast<const uint4*>(_weights.data_ptr<int>());  // weights is [OC, IC] but in FP6.
    auto in_feats = reinterpret_cast<const half*>(_in_feats.data_ptr<at::Half>());
    auto scales   = reinterpret_cast<const half*>(_scales.data_ptr<at::Half>());
    // Output Tensors
    auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
    at::Tensor _out_feats = torch::empty({num_in_feats, num_out_channels}, options);
    auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());

    options = torch::TensorOptions().dtype(torch::kFloat32).device(_in_feats.device());
    at::Tensor _workspace = torch::empty({splitK, num_in_feats, num_out_channels}, options);
    auto Reduction_Workspace = reinterpret_cast<float*>(_workspace.data_ptr<float>());  // Reduction_Workspace_Size = Split_K * M_Global * N_Global * sizeof(fp32)
      
    fp6_linear_kernel(0, // Using default stream here.
                      weight,
                      scales,
                      in_feats,
                      out_feats,
                      M,
                      N,
                      K, 
                      Reduction_Workspace,  
                      splitK);

    return _out_feats;
}


/*
 * Weight prepacking (Pytorch interface).
 * [Input & Output]
 *  fp6_tensor: int tensor of shape [OC, IC // 16 * 3];   // 3 INT32 words contains 16 FP6 weights.
 * [Output]
 *  packed_tensor: int tensor of shape [OC, IC // 16 * 3];
 */
torch::Tensor weight_matrix_prepacking_cpu(torch::Tensor fp6_tensor)
{
    size_t OC = fp6_tensor.size(0);
    size_t IC = fp6_tensor.size(1);
    assert (IC%3==0);   
    IC = IC*16/3;
    assert( (OC%256==0) && (IC%64==0) );
    auto packed_tensor = torch::empty_like(fp6_tensor);
    auto packed_tensor_ptr = reinterpret_cast<int*>(packed_tensor.data_ptr<int>());
    auto fp6_tensor_ptr = reinterpret_cast<int*>(fp6_tensor.data_ptr<int>());
    weight_matrix_prepacking(packed_tensor_ptr, fp6_tensor_ptr, OC, IC);
    return packed_tensor;
}

/*
 * Dequant a FP6 matrix to a equivalent FP16 matrix using CPUs.
 * A useful tool to construct input matrices for the FP16 GEMM baseline.
 * [Input]
 *  fp6_tensor:  int  tensor of shape [OC, IC // 16 * 3];   // 3 INT32 words contains 16 FP6  weights.
 *  fp16_scale:  half tensor of shape [OC];                 // for row-wise quantization.
 * [Output]
 *  fp16_tensor: half tensor of shape [OC, IC].     
 */
torch::Tensor weight_matrix_dequant_cpu(torch::Tensor fp6_tensor, torch::Tensor fp16_scale) 
{
    int OC = fp6_tensor.size(0);
    assert(fp6_tensor.size(1) % 3 == 0);
    int IC = fp6_tensor.size(1) / 3 * 16;
    assert(fp16_scale.size(0)==OC);
    //
    auto fp6_tensor_ptr = reinterpret_cast<int*>(fp6_tensor.data_ptr<int>());
    auto fp16_scale_ptr = reinterpret_cast<half*>(fp16_scale.data_ptr<at::Half>());
    //
    auto options = torch::TensorOptions().dtype(fp16_scale.dtype()).device(fp16_scale.device());
    at::Tensor fp16_tensor = torch::empty({OC, IC}, options);
    auto fp16_tensor_ptr = reinterpret_cast<half*>(fp16_tensor.data_ptr<at::Half>());
    //
    DeQuantMatrix_FP6_To_FP16(fp16_tensor_ptr, (unsigned char*)fp6_tensor_ptr, OC, IC, fp16_scale_ptr);
    //
    return fp16_tensor;
}

/////////////////////////////////////////////////// New Interface Supporting FPx /////////////////////////////////////////////////////////////////////
/*
Computes FPx-FP16 GEMM (PyTorch interface).

[Mathmatical Formula]
Standard definition of linear layer:    Out = In * trans(W), where In, Out, and W are stored in row-major.
After Equivalent transformation    :    trans(Out) = W * trans(In). Note that we do not perform "transpose" during runtime, we instead interpret the In/Out as column-major matrices when calling our CUDA kernel.

[Inputs]
  _in_feats:  tensor of shape [B, IC];                  // half 
  _weights:   int tensor of shape [OC, IC // 32 * x];   // x INT32 words contains 32 FPx weights.
  _scales:    tensor of shape [OC];                     // half
  splitK:     spliting the MatMul problem along K dimension for higher GPU utilization, default 1.
[Outputs]
  _out_feats: tensor of shape [B, OC];                  // half
*/
torch::Tensor fp_eXmY_linear_forward_cuda(
    int             EXPONENT,
    int             MANTISSA,
    torch::Tensor   _in_feats,
    torch::Tensor   _weights,
    torch::Tensor   _scales,
    int             splitK=1)
{
    int num_in_feats      = _in_feats.size(0);
    int num_in_channels   = _in_feats.size(1);
    int num_out_channels  = _weights.size(0);
    assert( num_in_channels%64 == 0 );
    assert( (num_in_channels/32*(1+EXPONENT+MANTISSA)) == _weights.size(1) );    // Making sure the K dimension is matched.
    //
    int M = num_out_channels;
    int K = num_in_channels;
    int N = num_in_feats;
    // Input Tensors
    auto weight = reinterpret_cast<const uint4*>(_weights.data_ptr<int>());  // weights is [OC, IC] but in FP6.
    auto in_feats = reinterpret_cast<const half*>(_in_feats.data_ptr<at::Half>());
    auto scales   = reinterpret_cast<const half*>(_scales.data_ptr<at::Half>());
    // Output Tensors
    auto options = torch::TensorOptions().dtype(_in_feats.dtype()).device(_in_feats.device());
    at::Tensor _out_feats = torch::empty({num_in_feats, num_out_channels}, options);
    auto out_feats = reinterpret_cast<half*>(_out_feats.data_ptr<at::Half>());

    options = torch::TensorOptions().dtype(torch::kFloat32).device(_in_feats.device());
    at::Tensor _workspace = torch::empty({splitK, num_in_feats, num_out_channels}, options);
    auto Reduction_Workspace = reinterpret_cast<float*>(_workspace.data_ptr<float>());  // Reduction_Workspace_Size = Split_K * M_Global * N_Global * sizeof(fp32)
    //
    fp_eXmY_linear_kernel(
        EXPONENT,
        MANTISSA,
        0, // Using default stream here.
        weight,
        scales,
        in_feats,
        out_feats,
        M,
        N,
        K, 
        Reduction_Workspace,  
        splitK);
    return _out_feats;
}


/*
 * Weight prepacking (Pytorch interface).
 * [Input & Output]
 *  fpx_tensor: int tensor of shape [OC, IC // 32 * x];
 * [Output]
 *  packed_tensor: int tensor of shape [OC, IC // 32 * x];
 */
torch::Tensor weight_matrix_prepacking_fp_eXmY_cpu(
    int EXPONENT,
    int MANTISSA,
    torch::Tensor fpx_tensor)
{
    int BIT_WIDTH = 1 + EXPONENT + MANTISSA;
    //
    size_t OC = fpx_tensor.size(0);
    size_t IC = fpx_tensor.size(1);
    assert (IC%BIT_WIDTH==0);   
    IC = IC*32/BIT_WIDTH;
    assert( (OC%256==0) && (IC%64==0) );
    auto packed_tensor = torch::empty_like(fpx_tensor);
    auto packed_tensor_ptr = reinterpret_cast<int*>(packed_tensor.data_ptr<int>());
    auto fpx_tensor_ptr = reinterpret_cast<int*>(fpx_tensor.data_ptr<int>());
    //
    weight_matrix_prepacking_fp_eXmY(EXPONENT, MANTISSA, packed_tensor_ptr, fpx_tensor_ptr, OC, IC);
    return packed_tensor;
}

/*
 * Dequant a FPx matrix to a equivalent FP16 matrix using CPUs.
 * A useful tool to construct input matrices for the FP16 GEMM baseline.
 * [Input]
 *  fpx_tensor:  int  tensor of shape [OC, IC // 32 * x];   //
 *  fp16_scale:  half tensor of shape [OC];                 // for row-wise quantization.
 * [Output]
 *  fp16_tensor: half tensor of shape [OC, IC].     
 */
torch::Tensor weight_matrix_dequant_fp_eXmY_cpu(
    int EXPONENT,
    int MANTISSA,
    torch::Tensor fpx_tensor,
    torch::Tensor fp16_scale) 
{
    int BIT_WIDTH = 1 + EXPONENT + MANTISSA;
    //
    int OC = fpx_tensor.size(0);
    assert(fpx_tensor.size(1) % BIT_WIDTH == 0);
    int IC = fpx_tensor.size(1) / BIT_WIDTH * 32;
    assert(fp16_scale.size(0)==OC);
    //
    auto fpx_tensor_ptr = reinterpret_cast<int*>(fpx_tensor.data_ptr<int>());
    auto fp16_scale_ptr = reinterpret_cast<half*>(fp16_scale.data_ptr<at::Half>());
    //
    auto options = torch::TensorOptions().dtype(fp16_scale.dtype()).device(fp16_scale.device());
    at::Tensor fp16_tensor = torch::empty({OC, IC}, options);
    auto fp16_tensor_ptr = reinterpret_cast<half*>(fp16_tensor.data_ptr<at::Half>());
    //
    dequant_matrix_fp_eXmY_to_fp16(EXPONENT, MANTISSA, fp16_tensor_ptr, (unsigned char*)fpx_tensor_ptr, OC, IC, fp16_scale_ptr);
    //
    return fp16_tensor;
}
#endif