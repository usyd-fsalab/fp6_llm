#include "hip/hip_runtime.h"
#include "kernel_test.h"
#include "fp6_linear.cuh"


int main(int argc, char** argv)
{
    // Parsing the inputs from CLI.
    if (argc != 7) {
        printf("Wrong Inputs! Correct input format: ./kernel_test EXPONENT MANTISSA #Row_Weight #Column_Weight BatchSize SplitK\n");
        return -1;
    }
    int EXPONENT    = atoi(argv[1]);
    int MANTISSA    = atoi(argv[2]);
    size_t M_GLOBAL = atoi(argv[3]);
    size_t K_GLOBAL = atoi(argv[4]);
    size_t N_GLOBAL = atoi(argv[5]);
    int    SPLIT_K  = atoi(argv[6]);
    int BIT_WIDTH = 1 + EXPONENT + MANTISSA;
    assert(EXPONENT==2 || EXPONENT==3);
    assert(MANTISSA==2);
    assert(M_GLOBAL%256==0);                 // Currently, M_GLOBAL must be a multiple of 256.
    assert(K_GLOBAL%64==0);                  // Currently, K_GLOBAL must be a multiple of 64.
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Matrices in quantized FPx models with faked values.
    unsigned char* A_xbit_h  = (unsigned char*)malloc(M_GLOBAL*K_GLOBAL*BIT_WIDTH/8);       CheckMallocCPU(A_xbit_h, __LINE__);     // Weight matrix with FP6 values, stored in row-major.
    for(size_t i=0; i<M_GLOBAL*K_GLOBAL*BIT_WIDTH/8; i++)   A_xbit_h[i] = rand() % 256;                                             // Random initialization.
    half*          A_Scale_h = (half*)malloc(M_GLOBAL*sizeof(half));                CheckMallocCPU(A_Scale_h, __LINE__);    // Quantization Scales with FP16 values.
    for(size_t i=0; i<M_GLOBAL; i++)                A_Scale_h[i] = float(rand()%256)/64.0f;                                 // Scale
    // Generaing FP16 format of the Weight Matrix
    half* A_16bit_h = (half*) malloc(M_GLOBAL*K_GLOBAL*sizeof(half));                           CheckMallocCPU(A_16bit_h, __LINE__);
    dequant_matrix_fp_eXmY_to_fp16(EXPONENT, MANTISSA, A_16bit_h, A_xbit_h, M_GLOBAL, K_GLOBAL, A_Scale_h);
    // In-place weight pre-packing
    weight_matrix_prepacking_fp_eXmY(EXPONENT, MANTISSA, (int*)A_xbit_h, (int*)A_xbit_h, M_GLOBAL, K_GLOBAL);

    // Devices Memory
    unsigned char*  A_xbit;
    half*           A_Scale;
    half*           A_16bit;
    hipMalloc(reinterpret_cast<void**>(&A_xbit),  M_GLOBAL*K_GLOBAL*BIT_WIDTH/8);             CheckMallocCUDA(A_xbit, __LINE__);
    hipMalloc(reinterpret_cast<void**>(&A_Scale), M_GLOBAL*sizeof(half));             CheckMallocCUDA(A_Scale, __LINE__);
    hipMalloc(reinterpret_cast<void**>(&A_16bit), M_GLOBAL*K_GLOBAL*sizeof(half));    CheckMallocCUDA(A_16bit, __LINE__);
    // Memory Copy from CPU to GPU
    hipMemcpy(A_xbit,     A_xbit_h,  M_GLOBAL*K_GLOBAL*BIT_WIDTH/8,          hipMemcpyHostToDevice);
    hipMemcpy(A_Scale,    A_Scale_h,          M_GLOBAL*sizeof(half),          hipMemcpyHostToDevice);
    hipMemcpy(A_16bit,             A_16bit_h,          M_GLOBAL*K_GLOBAL*sizeof(half), hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // B Matrix: Activations
    half* B_h = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL); CheckMallocCPU(B_h);       // col major 
    for (size_t i = 0; i < N_GLOBAL * K_GLOBAL; i++)
        B_h[i] = __float2half_rn(static_cast<float>((rand() % 5)) / 5 - 0.5f);
    // Device memory
    half* B            = NULL;
    hipMalloc(reinterpret_cast<void**>(&B), sizeof(half) * N_GLOBAL * K_GLOBAL);               CheckMallocCUDA(B, __LINE__);
    // Memory Copy from CPU to GPU
    hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipblasStatus_t cublas_status;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkLastCudaError(__LINE__);
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //printf("Launching CuBlas...\n");
    half* D_cublas = NULL;
    hipMalloc(reinterpret_cast<void**>(&D_cublas), sizeof(half) * M_GLOBAL * N_GLOBAL);        CheckMallocCUDA(D_cublas, __LINE__);
    hipMemset(D_cublas, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, 0);
    //hipblasSetMathMode(handle, HIPBLAS_PEDANTIC_MATH);          // Tensor core NOT enabled
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);             // Tensor core enabled
    hipDeviceSynchronize();
    int              m = M_GLOBAL, n = N_GLOBAL, k = K_GLOBAL;
    const float      alpha     = 1.0;
    const float      beta      = 0.0;
    hipblasGemmAlgo_t CuBlasALG = static_cast<hipblasGemmAlgo_t>(0);
    for (int i = 0; i < WARM_UP_ITERATION; i++) {
        cublas_status = hipblasGemmEx(handle,
                                     HIPBLAS_OP_T,   HIPBLAS_OP_N,
                                     m, n, k,
                                     &alpha,
                                     A_16bit,   HIP_R_16F, k,
                                     B,         HIP_R_16F, k,
                                     &beta,
                                     D_cublas,  HIP_R_16F, m,
                                     HIP_R_32F,
                                     CuBlasALG);
        checkCublasError(cublas_status, __LINE__);
    }
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        cublas_status = hipblasGemmEx(handle,
                                     HIPBLAS_OP_T,   HIPBLAS_OP_N,
                                     m, n, k,
                                     &alpha,
                                     A_16bit,   HIP_R_16F, k,
                                     B,         HIP_R_16F, k,
                                     &beta,
                                     D_cublas,  HIP_R_16F, m,
                                     HIP_R_32F,
                                     CuBlasALG);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //
    float milliseconds_cublas = 0;
    hipEventElapsedTime(&milliseconds_cublas, start, stop);
    milliseconds_cublas = milliseconds_cublas / BENCHMARK_ITERATION;
    float tflops_cublas = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_cublas / 1000.)) / 1e12;
    //
    half* D_cublas_h = NULL;  // col major
    D_cublas_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);   CheckMallocCPU(D_cublas_h);
    hipMemcpy(D_cublas_h, D_cublas, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_cublas);
    checkLastCudaError(__LINE__);
    /////////////////////////////////////////////////////////////////////////////////////////////////
    //printf("Launching FP6-LLM...\n");
    half* D_fp6 = NULL;
    hipMalloc(reinterpret_cast<void**>(&D_fp6), sizeof(half) * M_GLOBAL * N_GLOBAL); CheckMallocCUDA(D_fp6);
    hipMemset(D_fp6, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    //
    int Split_K = SPLIT_K;
    float* Reduction_Workspace = NULL;
    hipMalloc(reinterpret_cast<void**>(&Reduction_Workspace), sizeof(float) * M_GLOBAL * N_GLOBAL * Split_K);   CheckMallocCUDA(Reduction_Workspace, __LINE__);
    //
    for (int i = 0; i < WARM_UP_ITERATION; i++)
        fp_eXmY_linear_kernel(  
            EXPONENT,
            MANTISSA,
            0,
            (uint4*)A_xbit, A_Scale,
            B,
            D_fp6,
            M_GLOBAL, N_GLOBAL, K_GLOBAL,
            Reduction_Workspace,  
            Split_K);
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        fp_eXmY_linear_kernel(  
            EXPONENT,
            MANTISSA,
            0,
            (uint4*)A_xbit, A_Scale,
            B,
            D_fp6,
            M_GLOBAL, N_GLOBAL, K_GLOBAL,
            Reduction_Workspace,  
            Split_K);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    checkLastCudaError(__LINE__);
    //
    float milliseconds_fp6 = 0.0f;
    hipEventElapsedTime(&milliseconds_fp6, start, stop);
    milliseconds_fp6 = milliseconds_fp6 / BENCHMARK_ITERATION;
    float tflops_fp6 = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_fp6 / 1000.)) / 1e12;
    half* D_fp6_h = NULL;  // col major
    D_fp6_h       = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipMemcpy(D_fp6_h, D_fp6, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost);  // Col Major
    hipFree(D_fp6);
    hipFree(Reduction_Workspace);
    /////////////////////////////////////////////////////////////////////////////////////////////////
    double totalRelativeError_fp6  = ComputeTotalError(D_cublas_h, D_fp6_h, M_GLOBAL, N_GLOBAL);
    printf("************************************* ");
    printf("[%d-bit Weights, e%dm%d] M: %d N: %d K: %d SplitK: %d", BIT_WIDTH, EXPONENT, MANTISSA, M_GLOBAL, N_GLOBAL, K_GLOBAL, SPLIT_K);
    printf(" ************************************\n");
    PrintPerformance("cuBLAS", milliseconds_cublas, tflops_cublas, 0.0);
    PrintPerformance("quant_llm", milliseconds_fp6, tflops_fp6, totalRelativeError_fp6);
    //PrintMismatch("fp6", 100, 0.002, D_cublas_h, D_fp6_h, M_GLOBAL, N_GLOBAL);

    free(D_cublas_h);
    free(D_fp6_h);
    hipFree(B);
    return 0;
}
